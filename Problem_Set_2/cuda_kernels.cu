#include "hip/hip_runtime.h"
#include "cuda_kernels.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>
#include <algorithm>
#include <SFML/Graphics.hpp>

#include "cuda_kernels.h"
#include "Macros.h"

hipDeviceProp_t g_CudaDeviceProp;


void CUDADeviceQuery()
{
	printf(" **CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev, driverVersion = 0, runtimeVersion = 0;

	for (dev = 0; dev < deviceCount; ++dev)
	{
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

		// Console log
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
		printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);


#if CUDART_VERSION >= 5000
		// This is supported in CUDA 5.0 (runtime API device properties)
		printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
		printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);

		if (deviceProp.l2CacheSize)
		{
			printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
		}

#else
		// This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
		int memoryClock;
		getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
		printf("  Memory Clock rate:                             %.0f Mhz\n", memoryClock * 1e-3f);
		int memBusWidth;
		getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
		printf("  Memory Bus Width:                              %d-bit\n", memBusWidth);
		int L2CacheSize;
		getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

		if (L2CacheSize)
		{
			printf("  L2 Cache Size:                                 %d bytes\n", L2CacheSize);
		}

#endif

		printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
			deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
			deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
		printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
			deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
		printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
			deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);


		printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
		printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
		printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
		printf("  Warp size:                                     %d\n", deviceProp.warpSize);
		printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
		printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
		printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		printf("  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
		printf("  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
		printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
		printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
		printf("  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
		printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
		printf("  Alignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
		printf("  Device has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
		printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
		printf("  Device supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
		printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);

		const char *sComputeMode[] =
		{
			"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
			"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
			"Prohibited (no host thread can use ::hipSetDevice() with this device)",
			"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
			"Unknown",
			NULL
		};
		printf("  Compute Mode:\n");
		printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
	}
}

void CUDAInitDevice()
{
	int CudaDevice = 0;
	hipSetDevice(CudaDevice);
	hipGetDeviceProperties(&g_CudaDeviceProp, CudaDevice);
}

inline const char * GetCUDAError()
{
	hipError_t err = hipGetLastError();
	return hipGetErrorString( err );
}

void CreateAndSetDeviceData(sf::Uint8 *d_Data, const size_t SizeOfData)
{
	if ( hipSuccess != hipMalloc((void **)&d_Data, SizeOfData) )
    	printf( "Error in hipMalloc. %s!\n", GetCUDAError() );
	// if ( hipSuccess != hipMemset(d_Data, 13, SizeOfData) )
    // 	printf( "Error in hipMemset. %s!\n", GetCUDAError() );
}

void HostDeviceCopyOperation(void * h_Data, void * d_Data, size_t SizeOfData, const eHostDeviceCopyOperation operation)
{
	if (operation == eHostDeviceCopyOperation::HostToDevice)
	{
		if ( hipSuccess != hipMemcpy(d_Data, h_Data, SizeOfData, hipMemcpyHostToDevice) )
			printf( "Error in hipMemcpy host to device. %s!\n", GetCUDAError() );
	}
	else if (operation == eHostDeviceCopyOperation::DeviceToHost)
	{
		if ( hipSuccess != hipMemcpy(h_Data, d_Data, SizeOfData, hipMemcpyDeviceToHost) )
			printf( "Error in hipMemcpy device to host. %s!\n", GetCUDAError() );
	}
}

void DeviceFreeData(void *h_Data)
{
	hipFree(h_Data);
}

__global__ void kernel_FillPixels(unsigned char * Pixels, const size_t ImgWidth, const size_t ImgHeight)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	size_t j = blockDim.y * blockIdx.y + threadIdx.y;


    if(i >= ImgWidth || j >= ImgHeight)
        return;

	size_t Idx = j * ImgWidth + i;

	printf("%i", Pixels[Idx]);
    Pixels[Idx] = 255;
}

void CUDAFillPixels(sf::Uint8 *d_Pixels, const size_t ImgWidth, const size_t ImgHeight)
{
	size_t Lenght = std::max(ImgWidth, ImgHeight);

	size_t NumBlocks = ceil((Lenght * Lenght) / static_cast<Real>(g_CudaDeviceProp.maxThreadsPerBlock));
	//We need at least 1 block
	NumBlocks = (NumBlocks == 0) ? 1 : NumBlocks;

	dim3 ThreadsPerBlock(sqrt(g_CudaDeviceProp.maxThreadsPerBlock), sqrt(g_CudaDeviceProp.maxThreadsPerBlock));

	kernel_FillPixels <<< NumBlocks, ThreadsPerBlock >>>(d_Pixels, ImgWidth, ImgHeight);
	if ( hipSuccess != hipGetLastError() )
    	printf( "Error in kernel_FillPixels. %s!\n", GetCUDAError() );

	hipDeviceSynchronize();
}